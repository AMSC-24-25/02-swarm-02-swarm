
#include <hip/hip_runtime.h>
#ifdef ENABLE_CUDA


#include <curand_kernel.h>
#include <cmath>
#include "FireflyAlgorithm_Cuda.h"  // Declaration for host-callable wrapper

// Basic objective functions used for testing
// is kinda limit of CUDA (everytime you need a new function, you need to add here, for now)
__device__ double evalObjective(
    const double* x, int dimensions, int type
) {
    if (type == 0) { // Sphere function
        double sum = 0.0;
        for (int d = 0; d < dimensions; ++d)
            sum += x[d] * x[d];
        return sum;
    } else if (type == 1) { // Rastrigin function
        double A = 10.0, res = A * dimensions;
        for (int d = 0; d < dimensions; ++d)
            res += x[d]*x[d] - A * cos(2 * M_PI * x[d]);
        return res;
    } else if (type == 2) { // Rosenbrock function
        double sum = 0.0;
        for (int d = 0; d < dimensions - 1; ++d) {
            double t1 = (x[d + 1] - x[d] * x[d]);
            double t2 = (1 - x[d]);
            sum += 100 * t1 * t1 + t2 * t2;
        }
        return sum;
    }
    return 0.0; // fallback
}

// Kernel: evaluate objective function for each firefly
__global__ void evaluateBrightnessCUDA(
    double* positions, double* brightness,
    int numFireflies, int dimensions, int objectiveType
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numFireflies) return;
    brightness[i] = evalObjective(&positions[i * dimensions], dimensions, objectiveType);
}

// Kernel: update firefly positions using shared memory
__global__ void updateFirefliesCUDA_shared(
    double* positions, double* brightness,
    int numFireflies, int dimensions,
    double alpha, double beta, double gamma,
    unsigned int seed,
    double* new_positions
) {
    // Shared memory layout: [positions | brightness]
    extern __shared__ double shared[];
    double* shared_positions = shared;
    double* shared_brightness = (double*)&shared_positions[blockDim.x * dimensions];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numFireflies) return;

    // Setup per-thread RNG
    curandState state;
    curand_init(seed, i, 0, &state);

    // Init: copy current position to output buffer
    for (int d = 0; d < dimensions; ++d)
        new_positions[i * dimensions + d] = positions[i * dimensions + d];

    // Iterate over all blocks (tiling)
    for (int block_j = 0; block_j * blockDim.x < numFireflies; ++block_j) {
        int j_global = block_j * blockDim.x + threadIdx.x;

        // Load batch into shared memory
        if (j_global < numFireflies) {
            for (int d = 0; d < dimensions; ++d)
                shared_positions[threadIdx.x * dimensions + d] = positions[j_global * dimensions + d];
            shared_brightness[threadIdx.x] = brightness[j_global];
        } else {
            // Fill with dummy data
            for (int d = 0; d < dimensions; ++d)
                shared_positions[threadIdx.x * dimensions + d] = 0.0;
            shared_brightness[threadIdx.x] = 1e20; // high dummy value
        }

        __syncthreads();

        // Update logic for firefly i
        for (int j_local = 0; j_local < blockDim.x; ++j_local) {
            int j = block_j * blockDim.x + j_local;
            if (j >= numFireflies) continue;

            if (shared_brightness[j_local] < brightness[i]) {
                // Compute Euclidean distance r
                double r = 0.0;
                for (int d = 0; d < dimensions; ++d) {
                    double diff = positions[i * dimensions + d] - shared_positions[j_local * dimensions + d];
                    r += diff * diff;
                }
                r = sqrt(r);
                double beta_eff = beta * exp(-gamma * r * r);

                // Move firefly i towards j with attractiveness and noise
                for (int d = 0; d < dimensions; ++d) {
                    double randNoise = 2.0 * curand_uniform_double(&state) - 1.0;
                    double xi = positions[i * dimensions + d];
                    double xj = shared_positions[j_local * dimensions + d];
                    new_positions[i * dimensions + d] += beta_eff * (xj - xi) + alpha * randNoise;
                }
            }
        }

        __syncthreads();
    }
}

// Host-callable wrapper to evaluate brightness
extern "C" void launchEvaluateBrightnessCUDA(
    double* d_positions, double* d_brightness,
    int numFireflies, int dimensions, int objectiveType,
    int threadsPerBlock
) {
    int blocksPerGrid = (numFireflies + threadsPerBlock - 1) / threadsPerBlock;
    evaluateBrightnessCUDA<<<blocksPerGrid, threadsPerBlock>>>(
        d_positions, d_brightness, numFireflies, dimensions, objectiveType
    );
    cudaDeviceSynchronize();
}

// Host-callable wrapper to update fireflies
extern "C" void launchUpdateFirefliesCUDA(
    double* d_positions, double* d_brightness,
    int numFireflies, int dimensions,
    double alpha, double beta, double gamma,
    unsigned int seed,
    int threadsPerBlock
) {
    int blocksPerGrid = (numFireflies + threadsPerBlock - 1) / threadsPerBlock;
    size_t sharedMemSize = threadsPerBlock * (dimensions * sizeof(double) + sizeof(double));

    // Allocate temporary buffer
    double* d_new_positions;
    cudaMalloc(&d_new_positions, numFireflies * dimensions * sizeof(double));

    updateFirefliesCUDA_shared<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
        d_positions, d_brightness,
        numFireflies, dimensions,
        alpha, beta, gamma,
        seed,
        d_new_positions
    );

    // Copy back updated positions
    cudaMemcpy(d_positions, d_new_positions, numFireflies * dimensions * sizeof(double), cudaMemcpyDeviceToDevice);
    cudaFree(d_new_positions);
    cudaDeviceSynchronize();
}

#endif // ENABLE_CUDA
